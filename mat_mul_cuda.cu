
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
using namespace std;

constexpr size_t N_INPUTS = 128*1024;
constexpr size_t N_NODES = 1024*1024;

template<typename T>
T random(T range_from, T range_to) {
	std::random_device                  rand_dev;
	std::mt19937                        generator(rand_dev());
	std::uniform_real_distribution<T>    distr(range_from, range_to);
	return distr(generator);
}

__global__ void
dotprod(const float *const mat1, const float *const mat2, float *const dotprods) {

    // Use the thread ID as the node ID.
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    // mat2_node points to the beginning of the elements for this node (each thread handles a node).
    const float *const mat2_node = mat2 + id*N_INPUTS;

    // Compute dot product for this node.
    float dp = 0;
    for (size_t i = 0; i < N_INPUTS; i++) {
        dp += mat2_node[i]*mat1[i];
    }
    dotprods[id] = dp;
}

void initialize_vec(vector<float> &src)
{
    float a = 1.0;
    float b = 11.0;
    for(int i=0; i<src.size(); i++)
    {
        
        src[i] = random<float>(a, b);
    }
}

int main()
{
    vector<float> mat1(N_INPUTS);
    initialize_vec(mat1);

    vector<float> mat2 (N_INPUTS);
    initialize_vec(mat2);

    float *mat1_ptr;
    hipMalloc(&mat1_ptr, N_INPUTS*sizeof(float));
    hipMemcpy(mat1_ptr, mat1.data(), N_INPUTS*sizeof(float), hipMemcpyHostToDevice);

    float *mat2_ptr;
    hipMalloc(&mat2_ptr, N_INPUTS*sizeof(float));
    hipMemcpy(mat2_ptr, mat2.data(), N_INPUTS*sizeof(float), hipMemcpyHostToDevice);

    float *dotprod_ptr;
    hipMalloc(&dotprod_ptr, N_NODES*sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();
    dotprod<<<N_NODES/1024, 1024>>>(mat1_ptr, mat2_ptr, dotprod_ptr);
    if(hipDeviceSynchronize())
        cout<< "Synced!" <<endl;

    vector<float>res(N_NODES, 0.0);
    hipMemcpy(res.data(), dotprod_ptr, N_NODES*sizeof(float), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end-start);

    cout<< "Time Taken for CUDA Mat-Mul: " << duration.count() << " microseconds" <<endl;
}